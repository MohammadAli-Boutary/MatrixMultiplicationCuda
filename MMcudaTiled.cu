#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define shmem_size 16 * 16 * 4
#define TILE_SIZE 16

__global__ void matrixMul(int *a, int *b, int *c, int M, int N, int P) {
    __shared__ int A[TILE_SIZE][TILE_SIZE];
    __shared__ int B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int temp_sum = 0;

    for (int i = 0; i < (N + TILE_SIZE - 1) / TILE_SIZE; ++i) {
        if ((row < M) && (i * TILE_SIZE + threadIdx.x < N)) {
            A[threadIdx.y][threadIdx.x] = a[row * N + i * TILE_SIZE + threadIdx.x];
        } else {
            A[threadIdx.y][threadIdx.x] = 0;
        }

        if ((col < P) && (i * TILE_SIZE + threadIdx.y < N)) {
            B[threadIdx.y][threadIdx.x] = b[(i * TILE_SIZE + threadIdx.y) * P + col];
        } else {
            B[threadIdx.y][threadIdx.x] = 0;
        }

        __syncthreads();

        for (int j = 0; j < TILE_SIZE; j++) {
            temp_sum += A[threadIdx.y][j] * B[j][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < P) {
        c[row * P + col] = temp_sum;
    }
}


void fillMatrixRandom(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            matrix[i * cols + j] = rand() % 100;
        }
    }
}

void print_matrix(int *matrix, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%d\t", matrix[i * n + j]);
        }
        printf("\n");
    }
}


int main() {

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


   int n = 1000;
   int m = 800;
   int p = 1200;

   printf("Matrix sizes: A(%d x %d), B(%d x %d)\n", n, m, m, p);

   size_t bytes_a = n * m * sizeof(int);
   size_t bytes_b = m * p * sizeof(int);
   size_t bytes_c = n * p * sizeof(int);

   int *h_a, *h_b, *h_c;

   h_a = (int*) malloc(bytes_a);
   h_b = (int*) malloc(bytes_b);
   h_c = (int*) malloc(bytes_c);   

   int *d_a, *d_b, *d_c;

   hipMalloc(&d_a,bytes_a);
   hipMalloc(&d_b,bytes_b);
   hipMalloc(&d_c,bytes_c);
   
   fillMatrixRandom(h_a, n, m);
   fillMatrixRandom(h_b, m, p);

   hipMemcpy(d_a,h_a,bytes_a, hipMemcpyHostToDevice);
   hipMemcpy(d_b,h_b,bytes_b, hipMemcpyHostToDevice);

   int BLOCK_SIZE = 16;

   dim3 grid((p + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
   dim3 threads(BLOCK_SIZE,BLOCK_SIZE);

   hipEventRecord(start);

   matrixMul<<<grid,threads>>>(d_a,d_b,d_c,n,m,p);
   hipDeviceSynchronize();

   hipEventRecord(stop);
   hipEventSynchronize(stop);

   float milliseconds = 0;
   hipEventElapsedTime(&milliseconds, start, stop);

   float seconds = milliseconds / 1000.0;

   printf("Kernel execution time for tiled Matrix Multiplication: %f seconds\n", seconds);

   hipMemcpy(h_c,d_c,bytes_c,hipMemcpyDeviceToHost); 

   free(h_a);
   free(h_b);
   free(h_c);
   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);

   

   return 0; }
